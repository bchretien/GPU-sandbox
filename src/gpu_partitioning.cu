#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/partition.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>

const int N = 1024*1024;

struct is_positive
{
__host__ __device__
bool operator()(const int &x)
{
  return x >= 0;
}
};

struct my_compare
{
  __device__ __host__ bool operator()(const float x, const float y) const
  {
    return !((x<0.0f) && (y>0.0f));
  }
};

void print_vec(const thrust::host_vector<int>& v)
{
  for(size_t i = 0; i < min((int)v.size(), 10); i++)
    std::cout << "  " << v[i];
  std::cout << "\n";
}

void partition_test ()
{
  std::cout << "### PARTITION VERSION ###" << std::endl;

  thrust::host_vector<int> keyVec(N);
  thrust::host_vector<int> valVec(N);

  int sign = 1;
  for(int i = 0; i < N; ++i)
  {
    keyVec[i] = sign * i;
    valVec[i] = i;
    sign *= -1;
  }

  // Copy host to device
  thrust::device_vector<int> d_keyVec = keyVec;
  thrust::device_vector<int> d_valVec = valVec;

  std::cout << "Before:\n  keyVec = ";
  print_vec(keyVec);
  std::cout << "  valVec = ";
  print_vec(valVec);

  // Partition key-val on device
  thrust::partition(thrust::make_zip_iterator(thrust::make_tuple(d_keyVec.begin(), d_valVec.begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(d_keyVec.end(), d_valVec.end())),
                    d_keyVec.begin(),
                    is_positive());
                    
  // Copy result back to host
  keyVec = d_keyVec;
  valVec = d_valVec;

  std::cout << "After:\n  keyVec = ";
  print_vec(keyVec);
  std::cout << "  valVec = ";
  print_vec(valVec);
  std::cout << std::endl;
}

void sort_test ()
{
  std::cout << "### SORT VERSION ###" << std::endl;

  thrust::host_vector<int> keyVec(N);
  thrust::host_vector<int> valVec(N);

  int sign = 1;
  for(int i = 0; i < N; ++i)
  {
    keyVec[i] = sign * i;
    valVec[i] = i;
    sign *= -1;
  }
  
  // Copy host to device
  thrust::device_vector<int> d_keyVec = keyVec;
  thrust::device_vector<int> d_valVec = valVec;

  std::cout << "Before:\n  keyVec = ";
  print_vec(keyVec);
  std::cout << "  valVec = ";
  print_vec(valVec);

  // Sort key-val on device
  thrust::sort_by_key(d_keyVec.begin(), d_keyVec.end(),
                      d_valVec.begin(), my_compare());
                    
  // Copy result back to host
  keyVec = d_keyVec;
  valVec = d_valVec;

  std::cout << "After:\n  keyVec = ";
  print_vec(keyVec);
  std::cout << "  valVec = ";
  print_vec(valVec);
  std::cout << std::endl;
}

int main()
{
  sort_test();
  partition_test();
}
