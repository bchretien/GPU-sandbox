#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <time.h>

struct ElementWiseProductBasic : public thrust::binary_function<float2,float2,float2>
{
    __host__ __device__
    float2 operator()(const float2& v1, const float2& v2) const
    {
        float2 res;
        res.x = v1.x * v2.x - v1.y * v2.y;
        res.y = v1.x * v2.y + v1.y * v2.x;
        return res;
    }
};

/**
 * See: http://www.embedded.com/design/embedded/4007256/Digital-Signal-Processing-Tricks--Fast-multiplication-of-complex-numbers%5D
 */
struct ElementWiseProductModified : public thrust::binary_function<float2,float2,float2>
{
    __host__ __device__
    float2 operator()(const float2& v1, const float2& v2) const
    {
        float2 res;
        float a, b, c, d, k;
        a = v1.x;
        b = v1.y;
        c = v2.x;
        d = v2.y;
        k = a * (c + d);
        d =  d * (a + b);
        c =  c * (b - a);
        res.x = k -d;
        res.y = k + c;
        return res;
    }
};

int get_random_int(int min, int max)
{
    return min + (rand() % (int)(max - min + 1));
}

thrust::host_vector<float2> init_vector(const size_t N)
{
    thrust::host_vector<float2> temp(N);
    for(size_t i = 0; i < N; i++)
    {
        temp[i].x = get_random_int(0, 10);
        temp[i].y = get_random_int(0, 10);
    }
    return temp;
}

int main(void)
{
    const size_t N = 100000;
    const bool compute_basic_product    = true;
    const bool compute_modified_product = true;

    srand(time(NULL));

    thrust::host_vector<float2>   h_A = init_vector(N);
    thrust::host_vector<float2>   h_B = init_vector(N);
    thrust::device_vector<float2> d_A = h_A;
    thrust::device_vector<float2> d_B = h_B;

    thrust::host_vector<float2> h_result(N);
    thrust::host_vector<float2> h_result_modified(N);

    if (compute_basic_product)
    {
        thrust::device_vector<float2> d_result(N);

        thrust::transform(d_A.begin(), d_A.end(),
                          d_B.begin(), d_result.begin(),
                          ElementWiseProductBasic());
        h_result = d_result;
    }

    if (compute_modified_product)
    {
        thrust::device_vector<float2> d_result_modified(N);

        thrust::transform(d_A.begin(), d_A.end(),
                          d_B.begin(), d_result_modified.begin(),
                          ElementWiseProductModified());
        h_result_modified = d_result_modified;
    }

    std::cout << std::fixed;
    for (size_t i = 0; i < 4; i++)
    {
        float2 a = h_A[i];
        float2 b = h_B[i];

        std::cout << "(" << a.x << "," << a.y << ")";
        std::cout << " * ";
        std::cout << "(" << b.x << "," << b.y << ")";

        if (compute_basic_product)
        {
            float2 prod = h_result[i];
            std::cout << " = ";
            std::cout << "(" << prod.x << "," << prod.y << ")";
        }

        if (compute_modified_product)
        {
            float2 prod_modified = h_result_modified[i];
            std::cout << " = ";
            std::cout << "(" << prod_modified.x << "," << prod_modified.y << ")";
        }
        std::cout << std::endl;
    }   

    return 0;
}
