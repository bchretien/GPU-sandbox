#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "cuda_utils.h"

template <typename T>
__global__ void test_kernel(T* A)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    A[tid] = (T)tid;

    // This works
    assert (A[tid] >= 0);

    // This fails
    assert (A[tid]
            >= 0);
}


template <typename T>
void print_array(T* ar, uint N)
{
    for (uint i = 0; i < N; ++i)
        std::cout << ar[i] << " ";
    std::cout << std::endl;
}


template <typename T>
void run_kernel (uint N)
{
    // host array
    T* h_A;

    // device array
    T* d_A;

    // allocate device memory
    CUDA_SAFE_CALL (hipMalloc (&d_A, N * sizeof (T)));

    // allocate host memory
    h_A = (T*) malloc (N * sizeof (T));

    // initialize host data
    memset (h_A, 0, N * sizeof (T));

    // copy data to the device
    CUDA_SAFE_CALL (hipMemcpy (d_A, h_A, N * sizeof (T), hipMemcpyHostToDevice));

    // launch kernel
    dim3 grid_size, block_size;
    block_size.x = N;
    test_kernel<<<grid_size, block_size>>> (d_A);

    // copy result back to host
    CUDA_SAFE_CALL (hipMemcpy (h_A, d_A, N * sizeof (T), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL (hipDeviceSynchronize());

    print_array (h_A, N);

    // free device memory
    CUDA_SAFE_CALL (hipFree (d_A));

    // free host memory
    free (h_A);
}


int main()
{
    typedef int type_t;

    // vector size
    const uint N = 10;

    // run kernel
    run_kernel<type_t> (N);
}
