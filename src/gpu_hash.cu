
#include <hip/hip_runtime.h>
#include <string.h>
#include <iostream>
#include <stdio.h>

#define INIT_A 0x67452301
#define INIT_B 0xefcdab89
#define INIT_C 0x98badcfe
#define INIT_D 0x10325476

#define SQRT_2 0x5a827999
#define SQRT_3 0x6ed9eba1

#define CUDA_CHECK_ERROR()  __cuda_check_errors(__FILE__, __LINE__)
#define CUDA_SAFE_CALL(err) __cuda_safe_call(err, __FILE__, __LINE__)

inline void __cuda_check_errors(const char *filename, const int line_number)
{
    hipError_t err = hipDeviceSynchronize();
    if(err != hipSuccess)
    {
        printf("CUDA error %i at %s:%i: %s\n",
               err, filename, line_number, hipGetErrorString(err));
        exit(-1);
    }
}

inline void __cuda_safe_call(hipError_t err, const char *filename, const int line_number)
{
    if (err != hipSuccess)
    {
        printf("CUDA error %i at %s:%i: %s\n",
               err, filename, line_number, hipGetErrorString(err));
        exit(-1);
    }
}

__device__ void NTLM(char *, int, char*);
__device__ __constant__ char itoa16[17] = "0123456789ABCDEF";

__global__ void NTBruteforce(char *hex_format){
    char test[4] = {'t', 'h', 'e', 'n'};
    NTLM(test, 4, hex_format);      
}

__device__ void NTLM(char *key, int key_length, char *hex_format) {
    unsigned int nt_buffer[16] = { 0 };
    unsigned int output[4] = { 0 };

    // Globals for rounds
    unsigned int a = INIT_A;
    unsigned int b = INIT_B;
    unsigned int c = INIT_C;
    unsigned int d = INIT_D;

    // Prepare the string for hash calculation
    int i;
    int length = key_length;

    for (i = 0; i < length / 2; i++)
        nt_buffer[i] = key[2 * i] | (key[2 * i + 1] << 16);

    // Padding
    if (length % 2 == 1)
        nt_buffer[i] = key[length - 1] | 0x800000;
    else
        nt_buffer[i] = 0x80;

    // Put the length
    nt_buffer[14] = length << 4;

    // NTLM hash calculation

    /* Round 1 */
    a += (d ^ (b & (c ^ d))) + nt_buffer[0];
    a = (a << 3) | (a >> 29);
    d += (c ^ (a & (b ^ c))) + nt_buffer[1];
    d = (d << 7) | (d >> 25);
    c += (b ^ (d & (a ^ b))) + nt_buffer[2];
    c = (c << 11) | (c >> 21);
    b += (a ^ (c & (d ^ a))) + nt_buffer[3];
    b = (b << 19) | (b >> 13);

    a += (d ^ (b & (c ^ d))) + nt_buffer[4];
    a = (a << 3) | (a >> 29);
    d += (c ^ (a & (b ^ c))) + nt_buffer[5];
    d = (d << 7) | (d >> 25);
    c += (b ^ (d & (a ^ b))) + nt_buffer[6];
    c = (c << 11) | (c >> 21);
    b += (a ^ (c & (d ^ a))) + nt_buffer[7];
    b = (b << 19) | (b >> 13);

    a += (d ^ (b & (c ^ d))) + nt_buffer[8];
    a = (a << 3) | (a >> 29);
    d += (c ^ (a & (b ^ c))) + nt_buffer[9];
    d = (d << 7) | (d >> 25);
    c += (b ^ (d & (a ^ b))) + nt_buffer[10];
    c = (c << 11) | (c >> 21);
    b += (a ^ (c & (d ^ a))) + nt_buffer[11];
    b = (b << 19) | (b >> 13);

    a += (d ^ (b & (c ^ d))) + nt_buffer[12];
    a = (a << 3) | (a >> 29);
    d += (c ^ (a & (b ^ c))) + nt_buffer[13];
    d = (d << 7) | (d >> 25);
    c += (b ^ (d & (a ^ b))) + nt_buffer[14];
    c = (c << 11) | (c >> 21);
    b += (a ^ (c & (d ^ a))) + nt_buffer[15];
    b = (b << 19) | (b >> 13);

    /* Round 2 */
    a += ((b & (c | d)) | (c & d)) + nt_buffer[0] + SQRT_2;
    a = (a << 3) | (a >> 29);
    d += ((a & (b | c)) | (b & c)) + nt_buffer[4] + SQRT_2;
    d = (d << 5) | (d >> 27);
    c += ((d & (a | b)) | (a & b)) + nt_buffer[8] + SQRT_2;
    c = (c << 9) | (c >> 23);
    b += ((c & (d | a)) | (d & a)) + nt_buffer[12] + SQRT_2;
    b = (b << 13) | (b >> 19);

    a += ((b & (c | d)) | (c & d)) + nt_buffer[1] + SQRT_2;
    a = (a << 3) | (a >> 29);
    d += ((a & (b | c)) | (b & c)) + nt_buffer[5] + SQRT_2;
    d = (d << 5) | (d >> 27);
    c += ((d & (a | b)) | (a & b)) + nt_buffer[9] + SQRT_2;
    c = (c << 9) | (c >> 23);
    b += ((c & (d | a)) | (d & a)) + nt_buffer[13] + SQRT_2;
    b = (b << 13) | (b >> 19);

    a += ((b & (c | d)) | (c & d)) + nt_buffer[2] + SQRT_2;
    a = (a << 3) | (a >> 29);
    d += ((a & (b | c)) | (b & c)) + nt_buffer[6] + SQRT_2;
    d = (d << 5) | (d >> 27);
    c += ((d & (a | b)) | (a & b)) + nt_buffer[10] + SQRT_2;
    c = (c << 9) | (c >> 23);
    b += ((c & (d | a)) | (d & a)) + nt_buffer[14] + SQRT_2;
    b = (b << 13) | (b >> 19);

    a += ((b & (c | d)) | (c & d)) + nt_buffer[3] + SQRT_2;
    a = (a << 3) | (a >> 29);
    d += ((a & (b | c)) | (b & c)) + nt_buffer[7] + SQRT_2;
    d = (d << 5) | (d >> 27);
    c += ((d & (a | b)) | (a & b)) + nt_buffer[11] + SQRT_2;
    c = (c << 9) | (c >> 23);
    b += ((c & (d | a)) | (d & a)) + nt_buffer[15] + SQRT_2;
    b = (b << 13) | (b >> 19);

    /* Round 3 */
    a += (d ^ c ^ b) + nt_buffer[0] + SQRT_3;
    a = (a << 3) | (a >> 29);
    d += (c ^ b ^ a) + nt_buffer[8] + SQRT_3;
    d = (d << 9) | (d >> 23);
    c += (b ^ a ^ d) + nt_buffer[4] + SQRT_3;
    c = (c << 11) | (c >> 21);
    b += (a ^ d ^ c) + nt_buffer[12] + SQRT_3;
    b = (b << 15) | (b >> 17);

    a += (d ^ c ^ b) + nt_buffer[2] + SQRT_3;
    a = (a << 3) | (a >> 29);
    d += (c ^ b ^ a) + nt_buffer[10] + SQRT_3;
    d = (d << 9) | (d >> 23);
    c += (b ^ a ^ d) + nt_buffer[6] + SQRT_3;
    c = (c << 11) | (c >> 21);
    b += (a ^ d ^ c) + nt_buffer[14] + SQRT_3;
    b = (b << 15) | (b >> 17);

    a += (d ^ c ^ b) + nt_buffer[1] + SQRT_3;
    a = (a << 3) | (a >> 29);
    d += (c ^ b ^ a) + nt_buffer[9] + SQRT_3;
    d = (d << 9) | (d >> 23);
    c += (b ^ a ^ d) + nt_buffer[5] + SQRT_3;
    c = (c << 11) | (c >> 21);
    b += (a ^ d ^ c) + nt_buffer[13] + SQRT_3;
    b = (b << 15) | (b >> 17);

    a += (d ^ c ^ b) + nt_buffer[3] + SQRT_3;
    a = (a << 3) | (a >> 29);
    d += (c ^ b ^ a) + nt_buffer[11] + SQRT_3;
    d = (d << 9) | (d >> 23);
    c += (b ^ a ^ d) + nt_buffer[7] + SQRT_3;
    c = (c << 11) | (c >> 21);
    b += (a ^ d ^ c) + nt_buffer[15] + SQRT_3;
    b = (b << 15) | (b >> 17);

    output[0] = a + 0x67452301;
    output[1] = b + 0xefcdab89;
    output[2] = c + 0x98badcfe;
    output[3] = d + 0x10325476;

    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // Convert the hash to hex (for being readable)
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    for(i=0; i<4; i++)
    {
        int j = 0;
        unsigned int n = output[i];

        // Iterate the bytes of the integer
        for(; j<4; j++)
        {
            unsigned int convert = n % 256;
            hex_format[i * 8 + j * 2 + 1] = itoa16[convert % 16];
            convert = convert / 16;
            hex_format[i * 8 + j * 2 + 0] = itoa16[convert % 16];
            n = n / 256;
        }
    }       
}


int main()
{
    char* d_hex;
    char h_hex[33] = "";

    CUDA_SAFE_CALL(hipMalloc(&d_hex, 33 * sizeof(char)));

    NTBruteforce<<<1, 1>>>(d_hex);

    CUDA_CHECK_ERROR();

    CUDA_SAFE_CALL(hipMemcpy(h_hex, d_hex, 32 * sizeof(char), hipMemcpyDeviceToHost)); 
    CUDA_SAFE_CALL(hipFree(d_hex));
    
    h_hex[32] = '\0';
    std::cout << h_hex << std::endl;
}
